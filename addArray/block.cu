#include <hip/hip_runtime.h>
#include <stdio.h>

int N=512;

__global__ void add(int *a , int* b , int* c)
{
    c[blockIdx.x]=a[blockIdx.x]+b[blockIdx.x];
}

__global__ void addThread(int* a,int* b,int* c)
{
    c[threadIdx.x]=a[threadIdx.x]+b[threadIdx.x];
}

void rand_init(int* a,int N)
{
    int i=0;
    for(i=0; i<N; i++)
    {
        a[i]=rand()%100;
    }
    return;
}

int main(void)
{
    int *a,*b,*c;
    int *device_a , *device_b , *device_c;
    int size = N * sizeof(int);

    a=(int*)malloc(size);  rand_init(a,N);
    b=(int*)malloc(size);  rand_init(b,N);
    c=(int*)malloc(size);

    hipMalloc((void **)&device_a,size);
    hipMalloc((void **)&device_b,size);
    hipMalloc((void **)&device_c,size);

    hipMemcpy(device_a , a , size ,hipMemcpyHostToDevice);
    hipMemcpy(device_b , b , size ,hipMemcpyHostToDevice);

    add<<<N,1>>>(device_a,device_b,device_c);
    addThread<<<1,N>>>(device_a,device_b,device_c);

    hipMemcpy(c , device_c , size ,hipMemcpyDeviceToHost);

    int i;
    for(i=0; i<N; i++)
    {
        printf("%d ",c[i]);
    }
    printf("\n");

    free(a); free(b); free(c);
    hipFree(device_a); hipFree(device_b); hipFree(device_c);

    return 0;
}

