  
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add(int *a,int *b,int *c)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	c[index] = a[index] + b[index];
}
void rand_init(int* a,int N)
{
    int i=0;
    for(i=0; i<N; i++)
    {
        a[i]=rand()%100;
    }
    return;
}
#define N (2048*2048)
#define THREADS_PER_BLOCK 512
int main()
{
	int *a,*b,*c;
    int *device_a , *device_b , *device_c;
    int size = N * sizeof(int);

    a=(int*)malloc(size);  rand_init(a,N);
    b=(int*)malloc(size);  rand_init(b,N);
    c=(int*)malloc(size);

    hipMalloc((void **)&device_a,size);
    hipMalloc((void **)&device_b,size);
    hipMalloc((void **)&device_c,size);

    hipMemcpy(device_a , a , size ,hipMemcpyHostToDevice);
    hipMemcpy(device_b , b , size ,hipMemcpyHostToDevice);
    //run gpu
    printf("gpu started\n");
    add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(device_a, device_b, device_c);
    printf("gpu ended\n");
    hipMemcpy(c , device_c , size ,hipMemcpyDeviceToHost);

    /*int i;
    for(i=0; i<N; i++)
    {
        printf("%d ",c[i]);
    }
    printf("\n");*/

    free(a); free(b); free(c);
    hipFree(device_a); hipFree(device_b); hipFree(device_c);
	return 0;
}