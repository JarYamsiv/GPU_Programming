#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void add(int *a,int* b,int* c)
{
	*c= *a + *b;
}

int main(void)
{
	int a,b,c;
	int *device_a,*device_b,*device_c;

	int size = sizeof(int);

	hipMalloc((void **)&device_a,size);
	hipMalloc((void **)&device_b,size);
	hipMalloc((void **)&device_c,size);

	a=2; b=7;

	hipMemcpy(device_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(device_b, &b, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(device_a,device_b,device_c);

	hipMemcpy(&c, device_c, size, hipMemcpyDeviceToHost);

	printf("%d\n",c);

	hipFree(device_a); hipFree(device_b); hipFree(device_c);

	return 0;
}
